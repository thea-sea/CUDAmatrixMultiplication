#include "hip/hip_runtime.h"
/* Start Header *****************************************************************/

/*! \file kernel.cu

     \author Thea Sea, thea.sea, 2102348

     \par email: thea.sea@digipen.edu

     \date 19/2/2023

     \brief Copyright (C) 2024 DigiPen Institute of Technology.

  Reproduction or disclosure of this file or its contents without the prior written consent of DigiPen Institute of Technology is prohibited. */

/* End Header *******************************************************************/
/*
* Copyright 2024 Digipen.  All rights reserved.
*
* Please refer to the end user license associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms
* is strictly prohibited.
*
*/
#include <hip/hip_runtime_api.h>
#include "helper.h"
//#include <hip/hip_runtime.h> //FIX __global__ and __shared__ undefined
#include <> //fix blockIdx undefined


//P and M column-major, N row-major
__global__ void matrixMultiply(FLOAT_TYPE* P,       //<! [out] and mxn matrix
	                            const FLOAT_TYPE* M, //<! [in] an mxk matrix
	                            const FLOAT_TYPE* N, //<! [in] an kxn matrix
	                            const int m, const int n, const int k) 
{
	// Shared memory for tiling input N array
	__shared__ FLOAT_TYPE N_s[TILE_WIDTH_RATIO_K][TILE_WIDTH_N];

	//do NOT change aboove
	//your code here
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH_M + ty;
    int Col = bx * TILE_WIDTH_N + tx;

    FLOAT_TYPE P_reg[TILE_WIDTH_N];

    // Initialize P_reg to zero
    for (int i = 0; i < TILE_WIDTH_N; ++i) {
        P_reg[i] = 0.0;
    }

    // Loop over the input tiles
    for (int cnt = 0; cnt < (k - 1) / TILE_WIDTH_RATIO_K + 1; ++cnt) 
    {
        // Load the tile of N into shared memory
        int nIter = cnt * TILE_WIDTH_RATIO_K + tx;
        if (nIter < k) {
            N_s[ty][tx] = N[nIter * n + Col];
        }
        else {
            N_s[ty][tx] = 0.0;
        }

        __syncthreads();

        // Loop over elements inside the tile of N
        for (int i = 0; i < TILE_WIDTH_RATIO_K; ++i) {
            // Load tile of matrix M into register
            FLOAT_TYPE Mval = 0.0;
            if (Row < m && cnt * TILE_WIDTH_RATIO_K + i < k) {
                Mval = M[Row * k + cnt * TILE_WIDTH_RATIO_K + i];
            }

            // Loop over and update the output elements
            for (int j = 0; j < TILE_WIDTH_N; ++j) {
                if (Col + j < n) {
                    P_reg[j] += Mval * N_s[i][j];
                }
            }
        }

        __syncthreads();
    }

    // Store the output array variable to P elements
    for (int j = 0; j < TILE_WIDTH_N; ++j) {
        if (Row < m && Col + j < n) {
            P[Row * n + Col + j] = P_reg[j];
        }
    }
	
}

void matrixMultiplyGPU(FLOAT_TYPE* P,
	FLOAT_TYPE* M,
	FLOAT_TYPE* N,
	int numMRows,
	int numNColumns,
	int numMColumns)
{
	//@@ Initialize the grid and block dimensions here

	dim3 dimGrid((numMRows - 1) / TILE_WIDTH_M + 1, (numNColumns - 1) / TILE_WIDTH_N + 1);
	dim3 dimBlock(TILE_WIDTH_M, 1);
	matrixMultiply<<<dimGrid, dimBlock>>>(P, M, N, numMRows, numNColumns, numMColumns);

	getLastCudaError("matrixMultiply failed\n");
	hipDeviceSynchronize();
}
